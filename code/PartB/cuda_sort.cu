#include "hip/hip_runtime.h"
/**
 * @file cuda_sort.c
 * @brief 
 * parallel sample sort with CUDA
 * 
 * single thread block
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "../util/utils.h"
#include "../util/timer.h"
#define MAX __INT_MAX__
__host__ void readSorting(char *filename, int* n, int** a, int** oracle);
__host__ void checkSorting(int n, int* a, int* oracle);
__device__ int* realloc(int* old_a, int length, int new_length);
__device__ void swap(int* a, int* b);
__device__ int partition(int* a, int low, int high);
__device__  void qsort(int* arr, int l, int h);

__global__ void kernel_sort(int* a, int* np, int* output, int* shared){
  int nthds = blockDim.x;
  int n = *np;
  int my_rank = threadIdx.x;
  int* global_splitters = &shared[0];
  int* final_splitters = &shared[nthds*(nthds-1)];
  int* bucket_counts = &shared[nthds*nthds];
  int length = n / nthds;
  int* subarray = (int*) malloc(sizeof(int)*length);
  int i, count, d, r, par;
  /* cyclic assignment for splitter selection (pseudo random) */
  for (i = my_rank, count = 0; i < n; i+=nthds){
    if (count == length){
      subarray = realloc(subarray, length, length*2);
      length *= 2;
    }
    subarray[count++] = a[i];
  }
  qsort(subarray, 0, count-1);

  /* local splitter selection */
  int* local_splitters = (int*) malloc(sizeof(int) * (nthds-1));
  d = count / nthds;
  r = count % nthds;
  for (i = 1, par = 0; i < nthds; i++){
    par += (i < r ? d+1 : d); 
    local_splitters[i-1] = subarray[par];
  }
  free(subarray);

  /* broadcast local splitters in shared memory */
  int start_pos = (nthds-1)*my_rank;
  for (i = 0; i < nthds-1; i++){
    global_splitters[start_pos+i] = local_splitters[i];
  }
  free(local_splitters);
  __syncthreads();

  /* global splitter sort and final splitter selection in shared memory by Thread 0*/
  if (my_rank == 0){
    qsort(global_splitters, 0, (nthds-1)*nthds-1);
    d = nthds / 2;
    for (i = 0; i < nthds-1; i++){
      final_splitters[i] = global_splitters[d+i*nthds];
    }
  }
  __syncthreads();

  /* bucket sort */
  int bucket_low, bucket_high;
  bucket_low = my_rank == 0? -1 : final_splitters[my_rank-1];
  bucket_high = my_rank == nthds-1 ? MAX : final_splitters[my_rank];
  int my_bucket_size = n / nthds;
  int* my_bucket = (int*) malloc(sizeof(int) * my_bucket_size);
  for (i = 0, count = 0; i< n; i++){
    if (a[i] > bucket_low && a[i] <= bucket_high){
      if (count == my_bucket_size){
        my_bucket = realloc(my_bucket, my_bucket_size, my_bucket_size*2);
        my_bucket_size *=2;
      }
      my_bucket[count++] = a[i];
    }
  }
  /* broadcast bucket count in shared memory */
  bucket_counts[my_rank] = count;
  __syncthreads();

  /* write result to correct position */
  for (i = 0, start_pos = 0; i< my_rank; i++){
    start_pos += bucket_counts[i];
  }
  qsort(my_bucket, 0, count-1);
  for (i = 0; i < count; i++){
    output[start_pos+i] = my_bucket[i];
  }
  free(my_bucket);
}

int main(int argc, char* argv[]){
  int n, nthds;
  int* a, *oracle, *output;
  if (argc != 3){
    printf("./cuda_sort <num_threads> <input_filename>\n");
    exit(-1);
  }
  /* read from file */
  readSorting(argv[2], &n, &a, &oracle);

  nthds = atoi(argv[1]);
  if (n < (nthds*nthds)) {
    printf("sample sort: n > square(nthds)\n");
    exit(-1);
  }
  output = (int*) malloc(sizeof(int)*n);

  /* host and device memory allocation */
  int* a_device, *out_device, *n_device, *shared;
  hipMalloc(&a_device, sizeof(int)*n);
  hipMalloc(&out_device, sizeof(int)*n);
  hipMalloc(&n_device, sizeof(int));
  hipMalloc(&shared, sizeof(int)*nthds*(nthds+1));

  hipMemcpy(a_device, a, sizeof(int)*n, hipMemcpyHostToDevice);
  hipMemcpy(n_device, &n, sizeof(int), hipMemcpyHostToDevice);
  hipDeviceSetLimit(hipLimitMallocHeapSize, sizeof(int)*n*10);

  /* performance measurement */
  double start, end;
  GET_TIME(start);

  kernel_sort<<<1, nthds>>>(a_device, n_device, out_device, shared);
  hipDeviceSynchronize();

  /* correctness checking & timing output*/
  GET_TIME(end);
  hipMemcpy(output, out_device, sizeof(int)*n, hipMemcpyDeviceToHost);
  checkSorting(n, output, oracle);
  printf("computed in %.8fs\n", end-start);

  hipFree(a_device); hipFree(n_device); hipFree(out_device);
  hipFree(shared);
  free(a); free(oracle); free(output);
  return 0;
}

/* ************* */
__host__ void readSorting(char *filename, int* n, int** a, int** oracle){
  int i;
  FILE* fr = fopen(filename, "r");
  fscanf(fr, "%d", n);
  *a = (int*) malloc(sizeof(int)* *n);
  *oracle = (int*) malloc(sizeof(int)* *n);
  for (i =0; i< *n; i++){
    fscanf(fr, "%d", &(*a)[i]);
  }
  fclose(fr);

  char output_buf[50];
  sprintf(output_buf, "../testcases/output_sorting_%d", *n);
  fr = fopen(output_buf, "r");
  for (i = 0; i< *n; i++){
    fscanf(fr, "%d", &(*oracle)[i]);
  }
  fclose(fr);
}

__host__ void checkSorting(int n, int* a, int* oracle){
  int i;
  int incorrect = 0;
  for (i = 0; i < n; i++){
    if (a[i] != oracle[i]){
      // printf("a[%d]=%d vs %d\n", i, a[i], oracle[i]);
      incorrect = 1;
      break;
    }
  }
  if (incorrect) printf("Sorting incorrect\n");
  else printf("Sorting correct\n");
}

__device__ int* realloc(int* old_a, int length, int new_length){
  int* new_a = (int*) malloc (sizeof(int)*new_length);

  for (int i=0; i<length; i++){
    new_a[i] = old_a[i];
  }
  free(old_a);
  return new_a;
}

/* ************* */
/* cuda qsort */
__device__ void swap(int* a, int* b){
  int temp = *a;
  *a = *b;
  *b = temp;
}

__device__ int partition(int* a, int low, int high){
  int pi = a[high];
  int i = low-1;
  for (int j = low; j < high; j++){
    if (a[j] < pi){
      i++;
      swap(&a[i], &a[j]);
    }
  }
  swap(&a[i+1], &a[high]);
  return i+1;
}

__device__  void qsort(int* arr, int l, int h){
  int* stack = (int*) malloc(sizeof(int) * (h - l + 1));
  int top = -1;

  stack[++top] = l;
  stack[++top] = h;

  while (top >= 0) {
    h = stack[top--];
    l = stack[top--];

    int p = partition(arr, l, h);

    if (p - 1 > l) {
      stack[++top] = l;
      stack[++top] = p - 1;
    }

    if (p + 1 < h) {
      stack[++top] = p + 1;
      stack[++top] = h;
    }
  }
  free(stack);
}
/* ************* */